#include "hip/hip_runtime.h"
#include "hipDNN.h"
#include <cstdlib>
#include <cassert>
#include <iostream>
#include <fstream>
#include <sstream>
#include <math.h>

#define EXIT_WAIVED 0

#define cudaMemcpyHTD(dest, src, nBytes) hipMemcpy(dest, src, nBytes, hipMemcpyHostToDevice)
#define cudaMemcpyDTH(dest, src, nBytes) hipMemcpy(dest, src, nBytes, hipMemcpyDeviceToHost)
#define cudaMemcpyDTD(dest, src, nBytes) hipMemcpy(dest, src, nBytes, hipMemcpyDeviceToDevice)
#define value_type float
#define BLOCKSIZE 512

#define FatalError(s) {                                                \
    std::stringstream _where, _message;                                \
    _where << __FILE__ << ':' << __LINE__;                             \
    _message << std::string(s) + "\n" << __FILE__ << ':' << __LINE__;\
    std::cerr << _message.str() << "\nAborting...\n";                  \
    hipDeviceReset();                                                 \
    exit(EXIT_FAILURE);                                                \
}

#define checkCUDNN(status) {                                           \
    std::stringstream _error;                                          \
    if (status != HIPDNN_STATUS_SUCCESS) {                              \
      _error << "CUDNN failure: " << status;                           \
      FatalError(_error.str());                                        \
    }                                                                  \
}

#define checkCudaErrors(status) {                                      \
    std::stringstream _error;                                          \
    if (status != 0) {                                                 \
      _error << "Cuda failure: " << status;                            \
      FatalError(_error.str());                                        \
    }                                                                  \
}

__global__ void updateFilter(value_type *d_in, value_type *grad, value_type *msq, value_type alpha, value_type gamma, int n, int batchSize) {
	int idx = threadIdx.x + blockIdx.x*blockDim.x;
	if(idx>=n)
		return;
	value_type temp = grad[idx];
	msq[idx] = (1-gamma)*msq[idx] + gamma*temp*temp;
	if(msq[idx] > 0.0f) {
		d_in[idx] -= (alpha/(1.0*batchSize))*(temp/sqrt(msq[idx]));
	}
}

class Layer {
	public:
		value_type *h_data, *d_data;
		value_type *h_bias, *d_bias;
		value_type *d_msq, *d_grad;
		int inputs;
		int outputs;
		int kernelDim;
		int stride;
		value_type iRangeD, iRangeB;
		Layer(int inputs_, int outputs_, int kernelDim_, int stride_, value_type iRangeD_, value_type iRangeB_) {
			inputs = inputs_;
			outputs = outputs_;
			kernelDim = kernelDim_;
			stride = stride_;
			iRangeD = iRangeD_;
			iRangeB = iRangeB_;
		}
		~Layer() {
			delete[] h_data;
			delete[] h_bias;
			checkCudaErrors(hipFree(d_data));
			checkCudaErrors(hipFree(d_bias));
			checkCudaErrors(hipFree(d_msq));
			checkCudaErrors(hipFree(d_grad));
		}
		void randInit(value_type **h_dt, value_type **d_dt, int size, value_type irange) {
			int sizeInBytes = size*sizeof(value_type);
			*h_dt = new value_type[size];
			checkCudaErrors(hipMalloc(d_dt, sizeInBytes));
			for(int i = 0; i < size; ++i) {
				(*h_dt)[i] = irange*((value_type)(rand()))/((value_type)RAND_MAX) - irange/2.0;
			}
			checkCudaErrors(cudaMemcpyHTD(*d_dt, *h_dt, sizeInBytes));
		}
		void initData() {
			randInit(&h_data, &d_data, inputs*outputs*kernelDim*kernelDim, iRangeD);
			#ifdef TEST
				std::cout << "Layer initData: done!" << std::endl;
			#endif
		}
		void initBias() {
			randInit(&h_bias, &d_bias, outputs, iRangeB);
			#ifdef TEST
				std::cout << "Layer initBias: done!" << std::endl;
			#endif
		}
		void initMsq() {
			int size = inputs*outputs*kernelDim*kernelDim;
			int sizeInBytes = size*sizeof(value_type);
			checkCudaErrors(hipMalloc(&d_msq, sizeInBytes));
		}
		void initGrad() {
			int size = inputs*outputs*kernelDim*kernelDim;
			int sizeInBytes = size*sizeof(value_type);
			checkCudaErrors(hipMalloc(&d_grad, sizeInBytes));
		}
		void init() {
			initData();
			initBias();
			initMsq();
			initGrad();
			resetMsq();
			resetGrad();
		}
		void resetMsq() {
			int size = inputs*outputs*kernelDim*kernelDim;
			int sizeInBytes = size*sizeof(value_type);
			checkCudaErrors(hipMemset(d_msq, 0.0f, sizeInBytes));
		}
		void resetGrad() {
			int size = inputs*outputs*kernelDim*kernelDim;
			int sizeInBytes = size*sizeof(value_type);
			checkCudaErrors(hipMemset(d_grad, 0.0f, sizeInBytes));
		}
		void update(value_type alpha, value_type gamma, int batchSize) {
			int size = inputs*outputs*kernelDim*kernelDim;
			dim3 threadsPerBlock(BLOCKSIZE);
			dim3 numBlocks((size-1)/threadsPerBlock.x + 1);
			updateFilter<<<numBlocks, threadsPerBlock>>>(d_data, d_grad, d_msq, alpha, gamma, size, batchSize);
		}
};

void printDeviceVector(int size, value_type *d_vec) {
    value_type *vec;
    vec = new value_type[size];
    hipDeviceSynchronize();
    hipMemcpy(vec, d_vec, size*sizeof(value_type), hipMemcpyDeviceToHost);
    for (int i = 0; i < size; i++)
    {
        std::cout << vec[i] << " ";
    }
    std::cout << std::endl;
    delete[] vec;
}

void printHostVector(int size, value_type *h_vec) {
    for (int i = 0; i < size; i++)
    {
        std::cout << h_vec[i] << " ";
    }
    std::cout << std::endl;
}

class Network {
	private:
		hipdnnDataType_t dataType;
	    hipdnnTensorFormat_t tensorFormat;
	    hipdnnHandle_t cudnnHandle;
	    hipdnnTensorDescriptor_t srcTensorDesc, dstTensorDesc, biasTensorDesc, dataGradTensorDesc, diffTensorDesc;
	    hipdnnFilterDescriptor_t filterDesc, filterGradDesc;
	    hipdnnConvolutionDescriptor_t convDesc;
	    
	    void createHandles() {
			checkCUDNN(hipdnnCreate(&cudnnHandle));
			checkCUDNN(hipdnnCreateTensorDescriptor(&srcTensorDesc));
			checkCUDNN(hipdnnCreateTensorDescriptor(&dstTensorDesc));
			checkCUDNN(hipdnnCreateTensorDescriptor(&biasTensorDesc));
			checkCUDNN(hipdnnCreateTensorDescriptor(&dataGradTensorDesc));
			checkCUDNN(hipdnnCreateTensorDescriptor(&diffTensorDesc));
			checkCUDNN(hipdnnCreateFilterDescriptor(&filterDesc));
			checkCUDNN(hipdnnCreateFilterDescriptor(&filterGradDesc));
			checkCUDNN(hipdnnCreateConvolutionDescriptor(&convDesc));
	    }

	    void destroyHandles() {
	    	checkCUDNN(hipdnnDestroyConvolutionDescriptor(convDesc));
	        checkCUDNN(hipdnnDestroyFilterDescriptor(filterDesc));
	        checkCUDNN(hipdnnDestroyFilterDescriptor(filterGradDesc));
	        checkCUDNN(hipdnnDestroyTensorDescriptor(srcTensorDesc));
	        checkCUDNN(hipdnnDestroyTensorDescriptor(dstTensorDesc));
	        checkCUDNN(hipdnnDestroyTensorDescriptor(biasTensorDesc));
	        checkCUDNN(hipdnnDestroyTensorDescriptor(dataGradTensorDesc));
	        checkCUDNN(hipdnnDestroyTensorDescriptor(diffTensorDesc));
	        checkCUDNN(hipdnnDestroy(cudnnHandle));
	    }

	public:
		Network() {
			dataType = HIPDNN_DATA_FLOAT;
			tensorFormat = HIPDNN_TENSOR_NCHW;
			createHandles();
		}

		~Network() {
			destroyHandles();
		}
		void resize(int size, value_type **data) {
	        if(*data != NULL)
	        {
	            checkCudaErrors(hipFree(*data));
	        }
	        checkCudaErrors(hipMalloc(data, size*sizeof(value_type)));
	    }
	    void addBias(const hipdnnTensorDescriptor_t& dstTensorDesc, const Layer& layer, int n, int c, int h, int w, value_type *data) {
	        checkCUDNN(hipdnnSetTensor4dDescriptor(biasTensorDesc,
	                                                tensorFormat,
	                                                dataType,
	                                                n, c,
	                                                h,
	                                                w));
	        value_type alpha = value_type(1);
	        value_type beta  = value_type(1);
	        checkCUDNN(hipdnnAddTensor(cudnnHandle, CUDNN_ADD_SAME_C,
	                                      &alpha, biasTensorDesc,
	                                      layer.d_bias,
	                                      &beta,
	                                      dstTensorDesc,
	                                      data));
	    }
	    void convoluteForward(const Layer& conv,
                          int& n, int& c, int& h, int& w,
                          value_type* srcData, value_type** dstData) {
        hipdnnConvolutionFwdAlgo_t algo;

        checkCUDNN(hipdnnSetTensor4dDescriptor(srcTensorDesc,
                                                tensorFormat,
                                                dataType,
                                                n, c,
                                                h, w));

        checkCUDNN(hipdnnSetFilter4dDescriptor(filterDesc,
                                              dataType,
                                              conv.outputs,
                                              conv.inputs, 
                                              conv.kernelDim,
                                              conv.kernelDim));
 
        checkCUDNN(hipdnnSetConvolution2dDescriptor(convDesc,
                                                    0,0, // padding
                                                    1,1, // stride
                                                    conv.stride,conv.stride, // upscale
                                                    HIPDNN_CROSS_CORRELATION));	//OR HIPDNN_CONVOLUTION
        // find dimension of convolution output
        checkCUDNN(hipdnnGetConvolution2dForwardOutputDim(convDesc,
                                                srcTensorDesc,
                                                filterDesc,
                                                &n, &c, &h, &w));

        checkCUDNN(hipdnnSetTensor4dDescriptor(dstTensorDesc,
                                                tensorFormat,
                                                dataType,
                                                n, c,
                                                h,
                                                w));
        checkCUDNN(hipdnnGetConvolutionForwardAlgorithm(cudnnHandle,
                                                srcTensorDesc,
                                                filterDesc,
                                                convDesc,
                                                dstTensorDesc,
                                                HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST,
                                                0,
                                                &algo
                                                ));
        resize(n*c*h*w, dstData);
        size_t sizeInBytes=0;
        void* workSpace=NULL;
        checkCUDNN(hipdnnGetConvolutionForwardWorkspaceSize(cudnnHandle,
                                                srcTensorDesc,
                                                filterDesc,
                                                convDesc,
                                                dstTensorDesc,
                                                algo,
                                                &sizeInBytes));
        if (sizeInBytes!=0)
        {
          checkCudaErrors( hipMalloc(&workSpace,sizeInBytes) );
        }
        value_type alpha = value_type(1);
        value_type beta  = value_type(0);
        checkCUDNN( hipdnnConvolutionForward(cudnnHandle,
                                              &alpha,
                                              srcTensorDesc,
                                              srcData,
                                              filterDesc,
                                              conv.d_data,
                                              convDesc,
                                              algo,
                                              workSpace,
                                              sizeInBytes,
                                              &beta,
                                              dstTensorDesc,
                                              *dstData) );
        //addBias(dstTensorDesc, conv, c, *dstData); THIS CALL TO BE UNDERSTOOD AND CHANGED
        if (sizeInBytes!=0)
        {
          checkCudaErrors( hipFree(workSpace) );
        }
    }
    void activationForward(int n, int c, int h, int w, value_type* srcData, value_type** dstData)
    {
        resize(n*c*h*w, dstData);
        checkCUDNN(hipdnnSetTensor4dDescriptor(srcTensorDesc,
                                                tensorFormat,
                                                dataType,
                                                n, c,
                                                h,
                                                w));
        checkCUDNN(hipdnnSetTensor4dDescriptor(dstTensorDesc,
                                                tensorFormat,
                                                dataType,
                                                n, c,
                                                h,
                                                w));
        value_type alpha = value_type(1);
        value_type beta  = value_type(0);
        checkCUDNN(hipdnnActivationForward(cudnnHandle,
                                            HIPDNN_ACTIVATION_SIGMOID,
                                            &alpha,
                                            srcTensorDesc,
                                            srcData,
                                            &beta,
                                            dstTensorDesc,
                                            *dstData));    
    }
    void convoluteBacwardData(const Layer& conv,
                          int& nI, int& cI, int& hI, int& wI,
                          value_type* diffData,
                          int& nO, int& cO, int& hO, int& wO,
                          value_type** gradData) {
    	resize(nO*cO*hO*wO, gradData);
    	checkCUDNN(hipdnnSetTensor4dDescriptor(diffTensorDesc,
                                                tensorFormat,
                                                dataType,
                                                nI, cI,
                                                hI, wI));
    	checkCUDNN(hipdnnSetTensor4dDescriptor(dataGradTensorDesc,
                                                tensorFormat,
                                                dataType,
                                                nO, cO,
                                                hO, wO));
    	 checkCUDNN(hipdnnSetFilter4dDescriptor(filterDesc,
                                              dataType,
                                              conv.outputs,
                                              conv.inputs, 
                                              conv.kernelDim,
                                              conv.kernelDim));
 
        checkCUDNN(hipdnnSetConvolution2dDescriptor(convDesc,
                                                    0,0, // padding
                                                    1,1, // stride
                                                    conv.stride,conv.stride, // upscale
                                                    HIPDNN_CROSS_CORRELATION));	//OR HIPDNN_CONVOLUTION
        value_type alpha = value_type(1);
        value_type beta  = value_type(0);
        checkCUDNN(hipdnnConvolutionBackwardData(cudnnHandle,
        										&alpha,
        										filterDesc,
        										conv.d_data,
        										diffTensorDesc,
        										diffData,
        										convDesc,
        										&beta,
        										dataGradTensorDesc,
        										*gradData));
        nI = nO;
        cI = cO;
        hI = hO;
        wI = wO;
    }
    void convoluteBacwardFilter(const Layer& conv,
                          int& nI, int& cI, int& hI, int& wI,
                          value_type* srcData,
                          int& nO, int& cO, int& hO, int& wO,
                          value_type* diffData, value_type**gradData) {

    	checkCUDNN(hipdnnSetTensor4dDescriptor(srcTensorDesc,
                                                tensorFormat,
                                                dataType,
                                                nI, cI,
                                                hI, wI));
    	checkCUDNN(hipdnnSetTensor4dDescriptor(diffTensorDesc,
                                                tensorFormat,
                                                dataType,
                                                nO, cO,
                                                hO, wO));
    	 checkCUDNN(hipdnnSetFilter4dDescriptor(filterGradDesc,
                                              dataType,
                                              conv.outputs,
                                              conv.inputs, 
                                              conv.kernelDim,
                                              conv.kernelDim));
 
        checkCUDNN(hipdnnSetConvolution2dDescriptor(convDesc,
                                                    0,0, // padding
                                                    1,1, // stride
                                                    conv.stride,conv.stride, // upscale
                                                    HIPDNN_CROSS_CORRELATION));	//OR HIPDNN_CONVOLUTION
        value_type alpha = value_type(1);
        value_type beta  = value_type(1);	//accumulate filter gradients
        checkCUDNN(hipdnnConvolutionBackwardFilter(cudnnHandle,
        										&alpha,
        										srcTensorDesc,
        										srcData,
        										diffTensorDesc,
        										diffData,
        										convDesc,
        										&beta,
        										filterGradDesc,
        										*gradData));
    }
    void activationBackward(int& n, int& c, int& h, int& w,
                          value_type* srcData,
                          value_type* diffData, value_type* dstData, value_type**gradData) {

    	resize(n*c*h*w, gradData);
    	checkCUDNN(hipdnnSetTensor4dDescriptor(srcTensorDesc,
                                                tensorFormat,
                                                dataType,
                                                n, c,
                                                h, w));
    	checkCUDNN(hipdnnSetTensor4dDescriptor(diffTensorDesc,
                                                tensorFormat,
                                                dataType,
                                                n, c,
                                                h, w));
    	checkCUDNN(hipdnnSetTensor4dDescriptor(dstTensorDesc,
                                                tensorFormat,
                                                dataType,
                                                n, c,
                                                h, w));
    	checkCUDNN(hipdnnSetTensor4dDescriptor(dataGradTensorDesc,
                                                tensorFormat,
                                                dataType,
                                                n, c,
                                                h, w));
 
        value_type alpha = value_type(1);
        value_type beta  = value_type(0);
        checkCUDNN(hipdnnActivationBackward(cudnnHandle,
        										HIPDNN_ACTIVATION_SIGMOID,
        										&alpha,
        										srcTensorDesc,
        										srcData,
        										diffTensorDesc,
        										diffData,
        										dstTensorDesc,
        										dstData,
        										&beta,
        										dataGradTensorDesc,
        										*gradData));
    }
};

struct LayerDim{
	int x, y, z, w;
};

class CNN {
	private:
		int numNNLayer;
		int numFltrLayer;
		Layer **fltrLyr;
		LayerDim *nnLayerDim;
		float learnRate;
		Network *network;
		value_type *d_nn;
		value_type *qVals;
		value_type gamma;
		int miniBatchSize;
		int firstNNLayerUnits;
		int lastNNLayerUnits;
		int totalNNUnits;
		int totalFltrUnits;
	public:
		CNN(std::string x, float z, float gamma_) {
			std::ifstream nnConfig(x.c_str());
			nnConfig >> numNNLayer;
			
			numFltrLayer = numNNLayer - 1;
			fltrLyr = new Layer*[numFltrLayer];
			nnLayerDim = new LayerDim[numNNLayer];

			nnConfig >> nnLayerDim[0].x >> nnLayerDim[0].y >> nnLayerDim[0].z >> nnLayerDim[0].w;
			miniBatchSize = nnLayerDim[0].w;

			totalFltrUnits = 0;
			for(int i = 0; i < numFltrLayer; ++i) {
				int in, out, ker, stride;
				float irD, irB;
				nnConfig >> in >> out >> ker >> stride >> irD >> irB;
				totalFltrUnits += in*out*ker*ker;
				fltrLyr[i] = new Layer(in, out, ker, stride, irD, irB);
			}
			nnConfig.close();
			learnRate = z;
			gamma = gamma_;
			network = new Network();

			d_nn = NULL;
			#ifdef TEST
				std::cout << "CNN Constructor: done!" << std::endl;
			#endif
		}
		~CNN() {
			for(int i = 0; i < numFltrLayer; ++i) {
				delete fltrLyr[i];
			}
			delete[] fltrLyr;
			delete network;
			delete[] nnLayerDim;
			delete[] qVals;
			checkCudaErrors(hipFree(d_nn));
		}
		void init() {
			initLayers();
			forwardPropToGetDim();
			allocateNNMem();
			#ifdef TEST
				printGenAttr();
				printNNLayerDim();
				printFltrLayerAttr();
			#endif
		}
		void initLayers() {
			for(int i = 0; i < numFltrLayer; ++i) {
				fltrLyr[i]->init();
			}
			#ifdef TEST
				std::cout << "CNN initLayers: done!" << std::endl;
			#endif
		}
		void forwardPropToGetDim() {
			value_type *dstData = NULL, *srcData = NULL;
			int n = nnLayerDim[0].w, c = nnLayerDim[0].z, h = nnLayerDim[0].x, w = nnLayerDim[0].y;

			int inputSize = n*c*h*w;
			checkCudaErrors(hipMalloc(&srcData, inputSize*sizeof(value_type)));
       		checkCudaErrors(hipMemset(srcData, 0, inputSize*sizeof(value_type)));	//ZERO MEMSET

       		for(int i = 0; i < numFltrLayer; ++i) {
       			network->convoluteForward(*fltrLyr[i], n, c, h, w, srcData, &dstData);
       			nnLayerDim[i+1].w = n;
       			nnLayerDim[i+1].z = c;
       			nnLayerDim[i+1].x = h;
       			nnLayerDim[i+1].y = w;
       			network->activationForward(n, c, h, w, dstData, &srcData);
       		}
       		#ifdef TEST
       			std::cout << "Resulting Weights: " << std::endl;
       			printDeviceVector(n*c*h*w, srcData);
       		#endif
       		checkCudaErrors(hipFree(srcData));
        	checkCudaErrors(hipFree(dstData));
        	#ifdef TEST
				std::cout << "CNN forwardPropToGetDim: done!" << std::endl;
			#endif
		}
		//has to be called after dimensions are known
		void allocateNNMem() {
			totalNNUnits = 0;
			firstNNLayerUnits = nnLayerDim[0].x*nnLayerDim[0].y*nnLayerDim[0].z*nnLayerDim[0].w;
			for(int i = 0; i < numNNLayer; ++i) {
				int temp = nnLayerDim[i].x*nnLayerDim[i].y*nnLayerDim[i].z*nnLayerDim[i].w;
				totalNNUnits += temp;
				if(i == numNNLayer - 1)
					lastNNLayerUnits = temp;
			}
			checkCudaErrors(hipMalloc(&d_nn, totalNNUnits*sizeof(value_type)));
			qVals = new value_type[lastNNLayerUnits];
			#ifdef TEST
				std::cout << "CNN allocateNNMem: done!" << std::endl;
			#endif
		}
		//h_inpLayer must have firstNNLayerUnits in it
		void forwardProp(value_type *h_inpLayer) {
			value_type *dstData = NULL, *srcData = NULL;
			int n = nnLayerDim[0].w, c = nnLayerDim[0].z, h = nnLayerDim[0].x, w = nnLayerDim[0].y;
			
			int inputSize = n*c*h*w;
			assert(inputSize == firstNNLayerUnits);
			checkCudaErrors(hipMalloc(&srcData, inputSize*sizeof(value_type)) );
       		checkCudaErrors(cudaMemcpyHTD(srcData, h_inpLayer, inputSize*sizeof(value_type)));
       		//copy to d_nn
       		checkCudaErrors(cudaMemcpyHTD(d_nn, h_inpLayer, inputSize*sizeof(value_type)));

       		int tnnu = inputSize;
			for(int i = 0; i < numFltrLayer; ++i) {
				network->convoluteForward(*fltrLyr[i], n, c, h, w, srcData, &dstData);
				//may be some different modes and
				// no activation at some points 
				// utility can be added here
				network->activationForward(n, c, h, w, dstData, &srcData);
				//cpy to d_nn
				assert(n*c*h*w == nnLayerDim[i+1].x*nnLayerDim[i+1].y*nnLayerDim[i+1].z*nnLayerDim[i+1].w);
				checkCudaErrors(cudaMemcpyDTD(d_nn + tnnu, srcData, n*c*h*w*sizeof(value_type)));
				tnnu += n*c*h*w;
			}
			assert(tnnu == totalNNUnits);
			assert(n*c*h*w == lastNNLayerUnits);

	        checkCudaErrors(hipMemcpy(qVals, srcData, lastNNLayerUnits*sizeof(value_type), hipMemcpyDeviceToHost));
			
			#ifdef TEST
				printDeviceVector(n*c*h*w, srcData);
			#endif

			checkCudaErrors(hipFree(srcData));
        	checkCudaErrors(hipFree(dstData));
        	#ifdef TEST
				std::cout << "CNN forwardProp: done!" << std::endl;
			#endif
		}

		void backwardProp(value_type *h_err) {
			value_type *diffData = NULL, *gradData = NULL;
			int n = nnLayerDim[numNNLayer-1].w, c = nnLayerDim[numNNLayer-1].z, h = nnLayerDim[numNNLayer-1].x, w = nnLayerDim[numNNLayer-1].y;
			int nI, cI, hI, wI;
			int inputSize = n*c*h*w;
			assert(inputSize == lastNNLayerUnits);
			checkCudaErrors(hipMalloc(&diffData, inputSize*sizeof(value_type)) );
       		checkCudaErrors(cudaMemcpyHTD(diffData, h_err, inputSize*sizeof(value_type)));

       		//reset all fltr layer gradient
       		for(int i = 0; i < numFltrLayer; ++i)
       			fltrLyr[i]->resetGrad();

       		int tnnu = totalNNUnits - inputSize;
       		for(int i = numFltrLayer; i >= 1; --i) {
       			nI = nnLayerDim[i-1].w, cI = nnLayerDim[i-1].z, hI = nnLayerDim[i-1].x, wI = nnLayerDim[i-1].y;
       			network->activationBackward(n, c, h, w, d_nn + tnnu, diffData, d_nn + tnnu, &gradData);
       			network->convoluteBacwardFilter(*fltrLyr[i-1], nI, cI, hI, wI, d_nn + tnnu - nI*cI*hI*wI, n, c, h, w, gradData, &(fltrLyr[i-1]->d_grad));
       			if(i > 1) {
       				network->convoluteBacwardData(*fltrLyr[i-1], n, c, h, w, gradData, nI, cI, hI, wI, &diffData);
	       			tnnu -= n*c*h*w;	//here n,c,h,w <- nI, cI, hI, wI
	       			assert(n==nI && c==cI && h==hI && w==wI);
	       		}
       			
       		}
       		assert(tnnu == firstNNLayerUnits);
       		//update layers
       		for(int i = 0; i < numFltrLayer; ++i)
       			fltrLyr[i]->update(learnRate, gamma, miniBatchSize);
       		checkCudaErrors(hipFree(diffData));
        	checkCudaErrors(hipFree(gradData));
		}

		int argMaxQVal(int numAction) {
			assert(numAction == nnLayerDim[numNNLayer-1].z);
			int idx = 0;
			for(int i = 0; i < numAction; ++i) {
				if(qVals[idx] < qVals[i])
					idx = i;
			}
			return idx;
		}

		value_type* getQVals() {
			return qVals;
		}

		void printGenAttr() {
			std::cout << "Total NN Units: " << totalNNUnits << std::endl;
			std::cout << "Total Fltr Units: " << totalFltrUnits << std::endl;
			std::cout << "First layer NN Units: " << firstNNLayerUnits << std::endl;
			std::cout << "Last layer NN Units: " << lastNNLayerUnits << std::endl;
			std::cout << "Num NN Layers: " << numNNLayer << std::endl;
			std::cout << "Num fltr layers: " << numFltrLayer << std::endl;
			std::cout << "Learning Rate: " << learnRate << std::endl;
		}

		void printNNLayerDim() {
			std::cout << "NN Layer Dimensions" << std::endl;
			for (int i = 0; i < numNNLayer; ++i) {
				int n = nnLayerDim[i].w, c = nnLayerDim[i].z, h = nnLayerDim[i].x, w = nnLayerDim[i].y;
				std::cout << "w z x y == n c h w: " << i << " layer " << n << " " << c << " " << h << " " << w << std::endl;
			}
			#ifdef TEST
				std::cout << "CNN nnLayerDim: done!" << std::endl;
			#endif
		}

		void printFltrLayerAttr() {
			std::cout << "Filter Layer Attributes" << std::endl;
			for (int i = 0; i < numFltrLayer; ++i) {
				int k = fltrLyr[i]->outputs, c = fltrLyr[i]->inputs, c1 = fltrLyr[i]->kernelDim, h = fltrLyr[i]->stride;
				float w = fltrLyr[i]->iRangeD, u = fltrLyr[i]->iRangeB;
				std::cout << "out in ker stride rangeD range B: " << i << " layer " << k << " " << c << " " << c1 << " " <<  h << " " << w << " " << u << std::endl;
			}
			#ifdef TEST
				std::cout << "CNN nnLayerDim: done!" << std::endl;
			#endif
		}

		void printFltrLayer(int i) {
			std::cout << "Filter Layer " << i << std::endl;
			assert(i < numFltrLayer);
			printDeviceVector(fltrLyr[i]->inputs*fltrLyr[i]->outputs*fltrLyr[i]->kernelDim*fltrLyr[i]->kernelDim, fltrLyr[i]->d_data);
		}

		void printAllFltrLayer() {
			for(int i = 0; i < numFltrLayer; ++i) {
				printFltrLayer(i);
			}
		}

		void printFltrLayerGrad(int i) {
			std::cout << "Filter Layer Grad " << i << std::endl;
			assert(i < numFltrLayer);
			printDeviceVector(fltrLyr[i]->inputs*fltrLyr[i]->outputs*fltrLyr[i]->kernelDim*fltrLyr[i]->kernelDim, fltrLyr[i]->d_grad);
		}

		void printAllFltrLayerGrad() {
			for(int i = 0; i < numFltrLayer; ++i) {
				printFltrLayerGrad(i);
			}
		}

		void testForwardAndBackward() {
			int inputSize = nnLayerDim[0].x*nnLayerDim[0].y*nnLayerDim[0].z*nnLayerDim[0].w;
			value_type *testInput = new value_type[inputSize];
			for(int i = 0; i < inputSize; ++i) {
				testInput[i] = ((value_type)(rand()))/((value_type)(RAND_MAX));
			}
			std::cout << "Test Input" << std::endl;
			printHostVector(inputSize, testInput);
			std::cout << "Filter Layers" << std::endl;
			printAllFltrLayer();
			forwardProp(testInput);
			std::cout << "argMaxQVal is: " << argMaxQVal(nnLayerDim[numNNLayer-1].z) << std::endl;

			std::cout << "Backpropagation Started: " << std::endl;
			value_type *err = new value_type[lastNNLayerUnits];
			for(int i = 0; i < lastNNLayerUnits; ++i) {
				err[i] = -qVals[i]/2.0;
			}
			std::cout << "Errors: " << std::endl;
			printHostVector(lastNNLayerUnits, err);
			backwardProp(err);
			std::cout << "Fltr Layers gradients " << std::endl;
			printAllFltrLayerGrad();
			std::cout << "Fltr Layers after backpropagation" << std::endl;
			printAllFltrLayer();
			delete[] testInput;
			delete[] err;
		}
};

int main() {
CNN cnn("nnConfigTest", 0.3, 0.1);
cnn.init();
cnn.testForwardAndBackward();
return 0;
}



//ACTIVATION_RELU not working ... WHYYYYYYYYYYYYYYYYYYYYYYY


