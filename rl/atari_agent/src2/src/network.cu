//network.cu
#include "network.h"
#include "hipDNN.h"
#include "hipDNN.h"
    
void Network::createHandles() {
	checkCUDNN(hipdnnCreate(&cudnnHandle));
	checkCUDNN(hipdnnCreateTensorDescriptor(&srcTensorDesc));
	checkCUDNN(hipdnnCreateTensorDescriptor(&dstTensorDesc));
	checkCUDNN(hipdnnCreateTensorDescriptor(&biasTensorDesc));
	checkCUDNN(hipdnnCreateTensorDescriptor(&dataGradTensorDesc));
	checkCUDNN(hipdnnCreateTensorDescriptor(&diffTensorDesc));
	checkCUDNN(hipdnnCreateFilterDescriptor(&filterDesc));
	checkCUDNN(hipdnnCreateFilterDescriptor(&filterGradDesc));
	checkCUDNN(hipdnnCreateConvolutionDescriptor(&convDesc));
}

void Network::destroyHandles() {
	checkCUDNN(hipdnnDestroyConvolutionDescriptor(convDesc));
    checkCUDNN(hipdnnDestroyFilterDescriptor(filterDesc));
    checkCUDNN(hipdnnDestroyFilterDescriptor(filterGradDesc));
    checkCUDNN(hipdnnDestroyTensorDescriptor(srcTensorDesc));
    checkCUDNN(hipdnnDestroyTensorDescriptor(dstTensorDesc));
    checkCUDNN(hipdnnDestroyTensorDescriptor(biasTensorDesc));
    checkCUDNN(hipdnnDestroyTensorDescriptor(dataGradTensorDesc));
    checkCUDNN(hipdnnDestroyTensorDescriptor(diffTensorDesc));
    checkCUDNN(hipdnnDestroy(cudnnHandle));
}


Network::Network() {
	dataType = HIPDNN_DATA_FLOAT;
	tensorFormat = HIPDNN_TENSOR_NCHW;
	createHandles();
}

Network::~Network() {
	destroyHandles();
}
void Network::resize(int size, value_type **data) {
    if(*data != NULL)
    {
        checkCudaErrors(hipFree(*data));
    }
    checkCudaErrors(hipMalloc(data, size*sizeof(value_type)));
}
void Network::addBias(const hipdnnTensorDescriptor_t& dstTensorDesc, const Layer& layer, int n, int c, int h, int w, value_type *data) {
    checkCUDNN(hipdnnSetTensor4dDescriptor(biasTensorDesc,
                                            tensorFormat,
                                            dataType,
                                            n, c,
                                            h,
                                            w));
    value_type alpha = value_type(1);
    value_type beta  = value_type(1);
    checkCUDNN(hipdnnAddTensor(cudnnHandle, CUDNN_ADD_SAME_C,
                                  &alpha, biasTensorDesc,
                                  layer.d_bias,
                                  &beta,
                                  dstTensorDesc,
                                  data));
}
void Network::convoluteForward(const Layer& conv,
                  int& n, int& c, int& h, int& w,
                  value_type* srcData, value_type** dstData) {
    hipdnnConvolutionFwdAlgo_t algo;

    checkCUDNN(hipdnnSetTensor4dDescriptor(srcTensorDesc,
                                            tensorFormat,
                                            dataType,
                                            n, c,
                                            h, w));

    checkCUDNN(hipdnnSetFilter4dDescriptor(filterDesc,
                                          dataType,
                                          conv.outputs,
                                          conv.inputs, 
                                          conv.kernelDim,
                                          conv.kernelDim));

    checkCUDNN(hipdnnSetConvolution2dDescriptor(convDesc,
                                                0,0, // padding
                                                conv.stride,conv.stride, // stride
                                                1,1, // upscale
                                                HIPDNN_CROSS_CORRELATION));	//OR HIPDNN_CONVOLUTION
    // find dimension of convolution output
    checkCUDNN(hipdnnGetConvolution2dForwardOutputDim(convDesc,
                                            srcTensorDesc,
                                            filterDesc,
                                            &n, &c, &h, &w));

    checkCUDNN(hipdnnSetTensor4dDescriptor(dstTensorDesc,
                                            tensorFormat,
                                            dataType,
                                            n, c,
                                            h,
                                            w));
    checkCUDNN(hipdnnGetConvolutionForwardAlgorithm(cudnnHandle,
                                            srcTensorDesc,
                                            filterDesc,
                                            convDesc,
                                            dstTensorDesc,
                                            HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST,
                                            0,
                                            &algo
                                            ));
    resize(n*c*h*w, dstData);
    size_t sizeInBytes=0;
    void* workSpace=NULL;
    checkCUDNN(hipdnnGetConvolutionForwardWorkspaceSize(cudnnHandle,
                                            srcTensorDesc,
                                            filterDesc,
                                            convDesc,
                                            dstTensorDesc,
                                            algo,
                                            &sizeInBytes));
    if (sizeInBytes!=0)
    {
      checkCudaErrors(hipMalloc(&workSpace,sizeInBytes));
    }
    value_type alpha = value_type(1);
    value_type beta  = value_type(0);
    checkCUDNN( hipdnnConvolutionForward(cudnnHandle,
                                          &alpha,
                                          srcTensorDesc,
                                          srcData,
                                          filterDesc,
                                          conv.d_data,
                                          convDesc,
                                          algo,
                                          workSpace,
                                          sizeInBytes,
                                          &beta,
                                          dstTensorDesc,
                                          *dstData) );
    //addBias(dstTensorDesc, conv, c, *dstData); THIS CALL TO BE UNDERSTOOD AND CHANGED
    if (sizeInBytes!=0)
    {
      checkCudaErrors(hipFree(workSpace));
    }
}
void Network::activationForward(int n, int c, int h, int w, value_type* srcData, value_type** dstData)
{
    resize(n*c*h*w, dstData);
    checkCUDNN(hipdnnSetTensor4dDescriptor(srcTensorDesc,
                                            tensorFormat,
                                            dataType,
                                            n, c,
                                            h,
                                            w));
    checkCUDNN(hipdnnSetTensor4dDescriptor(dstTensorDesc,
                                            tensorFormat,
                                            dataType,
                                            n, c,
                                            h,
                                            w));
    value_type alpha = value_type(1);
    value_type beta  = value_type(0);
    checkCUDNN(hipdnnActivationForward(cudnnHandle,
                                        HIPDNN_ACTIVATION_RELU,
                                        &alpha,
                                        srcTensorDesc,
                                        srcData,
                                        &beta,
                                        dstTensorDesc,
                                        *dstData));    
}
void Network::convoluteBacwardData(const Layer& conv,
                      int& nI, int& cI, int& hI, int& wI,
                      value_type* diffData,
                      int& nO, int& cO, int& hO, int& wO,
                      value_type** gradData) {
	resize(nO*cO*hO*wO, gradData);
	checkCUDNN(hipdnnSetTensor4dDescriptor(diffTensorDesc,
                                            tensorFormat,
                                            dataType,
                                            nI, cI,
                                            hI, wI));
	checkCUDNN(hipdnnSetTensor4dDescriptor(dataGradTensorDesc,
                                            tensorFormat,
                                            dataType,
                                            nO, cO,
                                            hO, wO));
	 checkCUDNN(hipdnnSetFilter4dDescriptor(filterDesc,
                                          dataType,
                                          conv.outputs,
                                          conv.inputs, 
                                          conv.kernelDim,
                                          conv.kernelDim));

    checkCUDNN(hipdnnSetConvolution2dDescriptor(convDesc,
                                                0,0, // padding
                                                conv.stride,conv.stride, // stride
                                                1,1, // upscale
                                                HIPDNN_CROSS_CORRELATION));	//OR HIPDNN_CONVOLUTION
    value_type alpha = value_type(1);
    value_type beta  = value_type(0);
    checkCUDNN(hipdnnConvolutionBackwardData(cudnnHandle,
    										&alpha,
    										filterDesc,
    										conv.d_data,
    										diffTensorDesc,
    										diffData,
    										convDesc,
    										&beta,
    										dataGradTensorDesc,
    										*gradData));
    nI = nO;
    cI = cO;
    hI = hO;
    wI = wO;
}
void Network::convoluteBacwardFilter(const Layer& conv,
                      int& nI, int& cI, int& hI, int& wI,
                      value_type* srcData,
                      int& nO, int& cO, int& hO, int& wO,
                      value_type* diffData, value_type**gradData) {

	checkCUDNN(hipdnnSetTensor4dDescriptor(srcTensorDesc,
                                            tensorFormat,
                                            dataType,
                                            nI, cI,
                                            hI, wI));
	checkCUDNN(hipdnnSetTensor4dDescriptor(diffTensorDesc,
                                            tensorFormat,
                                            dataType,
                                            nO, cO,
                                            hO, wO));
	 checkCUDNN(hipdnnSetFilter4dDescriptor(filterGradDesc,
                                          dataType,
                                          conv.outputs,
                                          conv.inputs, 
                                          conv.kernelDim,
                                          conv.kernelDim));

    checkCUDNN(hipdnnSetConvolution2dDescriptor(convDesc,
                                                0,0, // padding
                                                conv.stride,conv.stride, // stride
                                                1,1, // upscale
                                                HIPDNN_CROSS_CORRELATION));	//OR HIPDNN_CONVOLUTION
    value_type alpha = value_type(1);
    value_type beta  = value_type(1);	//accumulate filter gradients
    checkCUDNN(hipdnnConvolutionBackwardFilter(cudnnHandle,
    										&alpha,
    										srcTensorDesc,
    										srcData,
    										diffTensorDesc,
    										diffData,
    										convDesc,
    										&beta,
    										filterGradDesc,
    										*gradData));
}
void Network::activationBackward(int& n, int& c, int& h, int& w,
                      value_type* srcData,
                      value_type* diffData, value_type* dstData, value_type**gradData) {

	resize(n*c*h*w, gradData);
	checkCUDNN(hipdnnSetTensor4dDescriptor(srcTensorDesc,
                                            tensorFormat,
                                            dataType,
                                            n, c,
                                            h, w));
	checkCUDNN(hipdnnSetTensor4dDescriptor(diffTensorDesc,
                                            tensorFormat,
                                            dataType,
                                            n, c,
                                            h, w));
	checkCUDNN(hipdnnSetTensor4dDescriptor(dstTensorDesc,
                                            tensorFormat,
                                            dataType,
                                            n, c,
                                            h, w));
	checkCUDNN(hipdnnSetTensor4dDescriptor(dataGradTensorDesc,
                                            tensorFormat,
                                            dataType,
                                            n, c,
                                            h, w));

    value_type alpha = value_type(1);
    value_type beta  = value_type(0);
    checkCUDNN(hipdnnActivationBackward(cudnnHandle,
    										HIPDNN_ACTIVATION_RELU,
    										&alpha,
    										srcTensorDesc,
    										srcData,
    										diffTensorDesc,
    										diffData,
    										dstTensorDesc,
    										dstData,
    										&beta,
    										dataGradTensorDesc,
    										*gradData));
}